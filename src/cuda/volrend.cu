#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <ctime>
#include <cstring>
#include <hip/hip_fp16.h>

#include "volrend/cuda/common.cuh"
#include "volrend/cuda/rt_core.cuh"
#include "volrend/render_options.hpp"
#include "volrend/internal/data_spec.hpp"
#include "volrend/render_context.hpp"

namespace volrend {

#define MAX3(a, b, c) max(max(a, b), c)
#define MIN3(a, b, c) min(min(a, b), c)

using internal::TreeSpec;
using internal::CameraSpec;

namespace {
template<typename scalar_t>
__host__ __device__ __inline__ static void screen2worlddir(
        int ix, int iy,
        const CameraSpec& cam,
        scalar_t* out,
        scalar_t* cen) {
    scalar_t xyz[3] ={ (ix - 0.5f * cam.width) / cam.fx,
                    -(iy - 0.5f * cam.height) / cam.fy, -1.0f};
    _mv3(cam.transform, xyz, out);
    _normalize(out);
    _copy3(cam.transform + 9, cen);
}
template<typename scalar_t>
__host__ __device__ __inline__ void maybe_world2ndc(
        const TreeSpec& tree,
        scalar_t* __restrict__ dir,
        scalar_t* __restrict__ cen) {
    if (tree.ndc_width <= 0)
        return;
    scalar_t t = -(1.f + cen[2]) / dir[2];
    for (int i = 0; i < 3; ++i) {
        cen[i] = cen[i] + t * dir[i];
    }

    dir[0] = -((2 * tree.ndc_focal) / tree.ndc_width) * (dir[0] / dir[2] - cen[0] / cen[2]);
    dir[1] = -((2 * tree.ndc_focal) / tree.ndc_height) * (dir[1] / dir[2] - cen[1] / cen[2]);
    dir[2] = -2 / cen[2];

    cen[0] = -((2 * tree.ndc_focal) / tree.ndc_width) * (cen[0] / cen[2]);
    cen[1] = -((2 * tree.ndc_focal) / tree.ndc_height) * (cen[1] / cen[2]);
    cen[2] = 1 + 2 / cen[2];

    _normalize(dir);
}

template<typename scalar_t>
__host__ __device__ __inline__ void rodrigues(
        const scalar_t* __restrict__ aa,
        scalar_t* __restrict__ dir) {
    scalar_t angle = _norm(aa);
    if (angle < 1e-6) return;
    scalar_t k[3];
    for (int i = 0; i < 3; ++i) k[i] = aa[i] / angle;
    scalar_t cos_angle = cos(angle), sin_angle = sin(angle);
    scalar_t cross[3];
    _cross3(k, dir, cross);
    scalar_t dot = _dot3(k, dir);
    for (int i = 0; i < 3; ++i) {
        dir[i] = dir[i] * cos_angle + cross[i] * sin_angle + k[i] * dot * (1.0 - cos_angle);
    }
}

template <typename scalar_t>
__host__ __device__ __inline__ static void world2screen(
    const CameraSpec& cam,
    const scalar_t* pos,
    scalar_t& x, scalar_t& y) {
    // matrix 4 x 3
    //  0   1   2
    //  3   4   5  
    //  6   7   8
    //  9  10  11 
    //transform[0] = v_right;
    //transform[1] = v_up;
    //transform[2] = v_back;
    //transform[3] = center;

    // world -> camera
    scalar_t w_xyz[3];
    w_xyz[0] = pos[0] - cam.transform[9];
    w_xyz[1] = pos[1] - cam.transform[10];
    w_xyz[2] = pos[2] - cam.transform[11];
    scalar_t c_x = _dot3(w_xyz, cam.transform);
    scalar_t c_y = _dot3(w_xyz, cam.transform + 3);
    scalar_t c_z = _dot3(w_xyz, cam.transform + 6);

    // camera -> image
    scalar_t c_z_inv = 1.0f / c_z;
    x = (cam.fx * c_x * c_z_inv + (cam.width >> 1)) - 0.5f;
    y = (cam.fy * c_y * c_z_inv + (cam.height >> 1)) - 0.5f;
}
template<typename scalar_t>
__host__ __device__ __inline__ scalar_t luminance(const scalar_t* rgb) {
    scalar_t coe[3] = {0.2126, 0.7152, 0.0722};
    return _dot3(rgb, coe);
}

}  // namespace

namespace device {

// Primary rendering kernel
__global__ static void render_kernel(
        hipSurfaceObject_t surf_obj,
        hipSurfaceObject_t surf_obj_depth,
        CameraSpec cam,
        TreeSpec tree,
        RenderOptions opt,
        float* probe_coeffs,
        RenderContext ctx, // use value, not reference
        bool offscreen) {
    CUDA_GET_THREAD_ID(idx, cam.width * cam.height);
    
    const int x = idx % cam.width, y = idx / cam.width;
    float dir[3], cen[3], out[4];
    uint8_t rgbx_init[4];
    if (!offscreen) {
        // Read existing values for compositing (with meshes)
        surf2Dread(reinterpret_cast<uint32_t*>(rgbx_init), surf_obj, x * 4,
                y, hipBoundaryModeZero);
    }

    bool enable_draw = tree.N > 0;
    out[0] = out[1] = out[2] = out[3] = 0.f;
    if (opt.enable_probe && y < opt.probe_disp_size + 5 &&
                            x >= cam.width - opt.probe_disp_size - 5) {
        // Draw probe circle
        float basis_fn[VOLREND_GLOBAL_BASIS_MAX];
        int xx = x - (cam.width - opt.probe_disp_size) + 5;
        int yy = y - 5;
        cen[0] = -(xx / (0.5f * opt.probe_disp_size) - 1.f);
        cen[1] = (yy / (0.5f * opt.probe_disp_size) - 1.f);

        float c = cen[0] * cen[0] + cen[1] * cen[1];
        if (c <= 1.f) {
            enable_draw = false;
            if (tree.data_format.basis_dim >= 0) {
                cen[2] = -sqrtf(1 - c);
                _mv3(cam.transform, cen, dir);

                internal::maybe_precalc_basis(tree, dir, basis_fn);
                for (int t = 0; t < 3; ++t) {
                    int off = t * tree.data_format.basis_dim;
                    float tmp = 0.f;
                    for (int i = opt.basis_minmax[0]; i <= opt.basis_minmax[1]; ++i) {
                        tmp += basis_fn[i] * probe_coeffs[off + i];
                    }
                    out[t] = 1.f / (1.f + expf(-tmp));
                }
                out[3] = 1.f;
            } else {
                for (int i = 0; i < 3; ++i)
                    out[i] = probe_coeffs[i];
                out[3] = 1.f;
            }
        } else {
            out[0] = out[1] = out[2] = 0.f;
        }
    }
    if (enable_draw) {
        screen2worlddir(x, y, cam, dir, cen);
        // out[3]=1.f;
        float vdir[3] = {dir[0], dir[1], dir[2]};
        maybe_world2ndc(tree, dir, cen);
        for (int i = 0; i < 3; ++i) {
            cen[i] = tree.offset[i] + tree.scale[i] * cen[i];
        }

        float t_max = 1e9f;
        if (!offscreen) {
            surf2Dread(&t_max, surf_obj_depth, x * sizeof(float), y, hipBoundaryModeZero);
        }

        rodrigues(opt.rot_dirs, vdir);

        if (opt.delta_tracking) {
            ctx.rng.advance(idx); // init random number generator
            const float dst = -__logf(1.0f - ctx.rng.next_float());
            delta_trace_ray(tree, dir, vdir, cen, opt, t_max, out, dst);
        } else {
            trace_ray(tree, dir, vdir, cen, opt, t_max, out);
        }
    }

    if (!opt.delta_tracking) {
        // Compositing with existing color
        const float nalpha = 1.f - out[3];
        if (offscreen) {
            const float remain = opt.background_brightness * nalpha;
            out[0] += remain;
            out[1] += remain;
            out[2] += remain;
        } else {
            out[0] += rgbx_init[0] / 255.f * nalpha;
            out[1] += rgbx_init[1] / 255.f * nalpha;
            out[2] += rgbx_init[2] / 255.f * nalpha;
        }

        // Output pixel color
        uint8_t rgbx[4] = {
            uint8_t(out[0] * 255),
            uint8_t(out[1] * 255),
            uint8_t(out[2] * 255),
            255
        };
        surf2Dwrite(
            *reinterpret_cast<uint32_t*>(rgbx),
            surf_obj,
            x * (int)sizeof(uint32_t),
            y,
            hipBoundaryModeZero); // squelches out-of-bound writes
    } else {
        // write float colors into delta tracking context
        surf2Dwrite(
            *reinterpret_cast<float4*>(out),
            ctx.surface[CUR_RGBA],
            x * (int)sizeof(float4),
            y,
            hipBoundaryModeZero); // squelches out-of-bound writes
    }
}

__global__ static void retrieve_cursor_lumisphere_kernel(
        TreeSpec tree,
        RenderOptions opt,
        float* out) {
    float cen[3];
    for (int i = 0; i < 3; ++i) {
        cen[i] = tree.offset[i] + tree.scale[i] * opt.probe[i];
    }

    float _cube_sz;
    const half* tree_val;
    internal::query_single_from_root(tree, cen, &tree_val, &_cube_sz);

    for (int i = 0; i < tree.data_dim - 1; ++i) {
        out[i] = __half2float(tree_val[i]);
    }
}

__global__ void temporal_accumulate(
        RenderContext ctx,
        RenderOptions opt,
        CameraSpec cam) {
    //const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    //if (n >= N)
    //    return;
    //uint32_t n_mul_3 = 3 * n;

    //// default: if no history, use current image
    //int n_prev = (int)n;
    //float count = 1;
    //float alpha = 1.0f;

    //float x = position[n_mul_3];
    //float y = position[n_mul_3 + 1];
    //float z = position[n_mul_3 + 2];
    //if (same_pose) {
    //    n_prev = n;
    //    count = history_count[n_prev] + 1;
    //    alpha = 1.0f / count;

    //}
    //else {
    //    // project to previous frame
    //    // world -> camera
    //    x -= poses_prev[3];
    //    y -= poses_prev[7];
    //    z -= poses_prev[11];
    //    float v_x = poses_prev[0] * x + poses_prev[4] * y + poses_prev[8] * z;
    //    float v_y = poses_prev[1] * x + poses_prev[5] * y + poses_prev[9] * z;
    //    float v_z = poses_prev[2] * x + poses_prev[6] * y + poses_prev[10] * z;
    //    // camera -> image
    //    float v_z_inv = 1.0f / v_z;
    //    float u_prev = (fx * v_x * v_z_inv + cx);
    //    float v_prev = (fy * v_y * v_z_inv + cy);

    //    // calculate pixel distance
    //    float u_cur = (n % width) + 0.5f;
    //    float v_cur = (n / width) + 0.5f;
    //    float du = u_cur - u_prev;
    //    float dv = v_cur - v_prev;
    //    if ((du * du + dv * dv) > 1.0f) {
    //        n_prev = floorf(v_prev) * width + floorf(u_prev);
    //    }
    //    else {
    //        n_prev = n;
    //    }

    //    if (n_prev < 0 || n_prev >= N) {
    //        n_prev = n;
    //    }

    //    // reset history count
    //    count = 4;
    //    // alpha = 1.0f / count;
    //    alpha = 0.2f;
    //}



    //// blend
    //history_count[n] = count;
    //depth_prev[n] = alpha * depth[n] + (1 - alpha) * depth_prev[n_prev];

    //uint32_t n_prev_mul_3 = 3 * n_prev;
    //image_prev[n_mul_3] = alpha * image[n_mul_3] + (1 - alpha) * image_prev[n_prev_mul_3];
    //image_prev[n_mul_3 + 1] = alpha * image[n_mul_3 + 1] + (1 - alpha) * image_prev[n_prev_mul_3 + 1];
    //image_prev[n_mul_3 + 2] = alpha * image[n_mul_3 + 2] + (1 - alpha) * image_prev[n_prev_mul_3 + 2];

    //// float c = (float)(n_prev == n);
    //// image_prev[n_mul_3] = c;
    //// image_prev[n_mul_3 + 1] = c;
    //// image_prev[n_mul_3 + 2] = c;
}


__global__ void wavelet_filter(
        RenderContext ctx,
        RenderOptions opt,
        int level) {
    //const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    //if (n >= N)
    //    return;

    constexpr static float epsilon = 1e-5f;
    constexpr static float h[25] = {
        1.0 / 256.0, 1.0 / 64.0, 3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
        1.0 / 64.0, 1.0 / 16.0, 3.0 / 32.0, 1.0 / 16.0, 1.0 / 64.0,
        3.0 / 128.0, 3.0 / 32.0, 9.0 / 64.0, 3.0 / 32.0, 3.0 / 128.0,
        1.0 / 64.0, 1.0 / 16.0, 3.0 / 32.0, 1.0 / 16.0, 1.0 / 64.0,
        1.0 / 256.0, 1.0 / 64.0, 3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0 };
    constexpr static float gaussianKernel[9] = {
        1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
        1.0 / 8.0, 1.0 / 4.0, 1.0 / 8.0,
        1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0 };

    // locate
    // const int x = n % width;
    // const int y = n / width;
    //scalar_t* pImage_ptr = image + n * 3;
    //scalar_t* pVariance_ptr = variance + n;
    //const scalar_t* pPosition_ptr = position + n * 3;
    //const scalar_t* pDensity_ptr = density + n;

    //const float pLuminance = pImage_ptr[0] * 0.2126f + pImage_ptr[1] * 0.7152f + pImage_ptr[2] * 0.0722f;
    //const float pVariance = pVariance_ptr[0];
    //const float pPosition_x = pPosition_ptr[0];
    //const float pPosition_y = pPosition_ptr[1];
    //const float pPosition_z = pPosition_ptr[2];
    //const float pDensity = pDensity_ptr[0];

    //// filter variance
    //int delta_locs[9] = {
    //    -width - 1, -width, -width + 1,
    //    -1, 0, 1,
    //    width - 1, width, width + 1,
    //};
    //float gaussian_sum = 0.0f;
    //float gaussian_sumw = 0.0f;
    //for (int i = 0; i < 9; i++) {
    //    int loc = (int)n + delta_locs[i];
    //    if (loc < 0 || loc >= N) continue;
    //    gaussian_sum += gaussianKernel[i] * variance[loc];
    //    gaussian_sumw += gaussianKernel[i];
    //}
    //float gaussian_v = gaussian_sumw > epsilon ? gaussian_sum / gaussian_sumw
    //    : 0;
    //const int x_step = 1 << level;
    //const int y_step = x_step * width;
    //const int northwest = (int)n - support * (x_step + y_step);
    //float r = 0.0f;
    //float g = 0.0f;
    //float b = 0.0f;
    //float v = 0.0f;
    //float weights = 0.0f;
    //float weight_squares = 0.0f;

    //for (int offsety = -support; offsety <= support; offsety++)
    //{
    //    int loc = northwest + (offsety - support) * y_step;
    //    for (int offsetx = -support; offsetx <= support; offsetx++, loc += x_step) {
    //        if (loc < 0 || loc >= N)
    //            continue;

    //        // locate
    //        const scalar_t* qImage_ptr = image + loc * 3;
    //        const scalar_t* qVariance_ptr = variance + loc;
    //        const scalar_t* qPosition_ptr = position + loc * 3;
    //        const scalar_t* qDensity_ptr = density + loc;

    //        const float qLuminance = qImage_ptr[0] * 0.2126f + qImage_ptr[1] * 0.7152f + qImage_ptr[2] * 0.0722f;
    //        const float qVariance = qVariance_ptr[0];
    //        const float qPosition_x = qPosition_ptr[0];
    //        const float qPosition_y = qPosition_ptr[1];
    //        const float qPosition_z = qPosition_ptr[2];
    //        const float qDensity = qDensity_ptr[0];

    //        float t_x = pPosition_x - qPosition_x;
    //        float t_y = pPosition_y - qPosition_y;
    //        float t_z = pPosition_z - qPosition_z;
    //        float dist_p = t_x * t_x + t_y * t_y + t_z * t_z;
    //        float wp = fminf(__expf(-dist_p / (kp + epsilon)), 1.0f);

    //        float dist_d = fabsf(pDensity - qDensity);
    //        float wd = fminf(__expf(-dist_d / (kd + epsilon)), 1.0f);

    //        float dist_l = fabsf(pLuminance - qLuminance);
    //        float wl = fminf(__expf(-dist_l / (kl * sqrtf(gaussian_v) + epsilon)), 1.0f);

    //        float w = wp * wd * wl;
    //        float weight = h[5 * (offsety + support) + offsetx + support] * w;

    //        float weight_square = weight * weight;
    //        weights += weight;
    //        weight_squares += weight_square;
    //        r += weight * qImage_ptr[0];
    //        g += weight * qImage_ptr[1];
    //        b += weight * qImage_ptr[2];
    //        v += weight_square * qVariance;
    //    }
    //}

    //if (weights > epsilon)
    //{
    //    pImage_ptr[0] = clamp(r / weights, 0.0f, 10.0f);
    //    pImage_ptr[1] = clamp(g / weights, 0.0f, 10.0f);
    //    pImage_ptr[2] = clamp(b / weights, 0.0f, 10.0f);
    //    pVariance_ptr[0] = fminf(fmaxf(v / (weights * weights), 0.0f), 10.0f);
    //}
}

__global__ void resultFromContext(
        hipSurfaceObject_t surf_obj,
        hipSurfaceObject_t surf_obj_depth,
        RenderContext ctx,
        RenderOptions opt,
        bool offscreen) {
    const int& width = ctx.prev_cam.width;
    const int& height = ctx.prev_cam.height;
    CUDA_GET_THREAD_ID(idx, width * height);
    const int x = idx % width, y = idx / width;

    uint8_t rgbx_init[4];
    if (!offscreen) {
        // Read existing values for compositing (with meshes)
        surf2Dread(
            reinterpret_cast<uint32_t*>(rgbx_init), 
            surf_obj, 
            x * (int)sizeof(uint32_t),
            y, 
            hipBoundaryModeZero);
    }

    // Compositing with existing color
    float out[4];
    surf2Dread(
        reinterpret_cast<float4*>(out),
        ctx.surface[CUR_RGBA], 
        x * (int)sizeof(float4),
        y, 
        hipBoundaryModeZero);
    const float nalpha = 1.f - out[3];
    if (offscreen) {
        const float remain = opt.background_brightness * nalpha;
        out[0] += remain;
        out[1] += remain;
        out[2] += remain;
    }
    else {
        out[0] += rgbx_init[0] / 255.f * nalpha;
        out[1] += rgbx_init[1] / 255.f * nalpha;
        out[2] += rgbx_init[2] / 255.f * nalpha;
    }

    // Output pixel color
    uint8_t rgbx[4] = {
        uint8_t(out[0] * 255),
        uint8_t(out[1] * 255),
        uint8_t(out[2] * 255),
        255
    };
    surf2Dwrite(
        *reinterpret_cast<uint32_t*>(rgbx),
        surf_obj,
        x * (int)sizeof(uint32_t),
        y,
        hipBoundaryModeZero); // squelches out-of-bound writes
}

}  // namespace device

__host__ void launch_renderer(const N3Tree& tree,
        const Camera& cam, const RenderOptions& options, hipArray_t& image_arr,
        hipArray_t& depth_arr,
        hipStream_t stream,
        RenderContext& ctx,
        bool offscreen) {
    hipSurfaceObject_t surf_obj = 0, surf_obj_depth = 0;

    float* probe_coeffs = nullptr;
    if (options.enable_probe) {
        cuda(Malloc(&probe_coeffs, (tree.data_dim - 1) * sizeof(float)));
        device::retrieve_cursor_lumisphere_kernel<<<1, 1, 0, stream>>>(
                tree,
                options,
                probe_coeffs);
    }

    {
        struct hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(res_desc));
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = image_arr;
        hipCreateSurfaceObject(&surf_obj, &res_desc);
    }
    if (!offscreen) {
        {
            struct hipResourceDesc res_desc;
            memset(&res_desc, 0, sizeof(res_desc));
            res_desc.resType = hipResourceTypeArray;
            res_desc.res.array.array = depth_arr;
            hipCreateSurfaceObject(&surf_obj_depth, &res_desc);
        }
    }
    if (options.delta_tracking) {
        ctx.createSurfaceObjects();
    }

    // less threads is weirdly faster for me than 1024
    // Not sure if this scales to a good GPU
    const int N_CUDA_THREADS = 512;
    const int blocks = N_BLOCKS_NEEDED(cam.width * cam.height, N_CUDA_THREADS);
    device::render_kernel<<<blocks, N_CUDA_THREADS, 0, stream>>>(
            surf_obj,
            surf_obj_depth,
            cam,
            tree,
            options,
            probe_coeffs,
            ctx,
            offscreen
    );
    if (options.delta_tracking) {
        // ===== denoise =====
        // temporal 
        device::temporal_accumulate<<<blocks, N_CUDA_THREADS, 0, stream>>>(
            ctx,
            options,
            cam
        );
        // record camera
        if (!ctx.has_history) {
            ctx.prev_cam.width = cam.width;
            ctx.prev_cam.height = cam.height;
            ctx.prev_cam.fx = cam.fx;
            ctx.prev_cam.fy = cam.fy;
        }
        cuda(Memcpy(ctx.prev_cam.transform, cam.device.transform,
            12 * sizeof(float), hipMemcpyDeviceToDevice
        ));
        
        // spatial
        for (int level = 0; level < options.filter_iters; level++) {
            device::wavelet_filter<<<blocks, N_CUDA_THREADS, 0, stream>>>(
                ctx,
                options,
                level
            );
        }

        // convert float rgb image to uint32_t rgb image
        device::resultFromContext<<<blocks, N_CUDA_THREADS, 0, stream>>>(
            surf_obj,
            surf_obj_depth,
            ctx,
            options,
            offscreen
        );

        // update context
        ctx.rng.advance();
        if (!ctx.has_history) {
            ctx.has_history = true;
        }
    }

    if (options.enable_probe) {
        hipFree(probe_coeffs);
    }
}
}  // namespace volrend

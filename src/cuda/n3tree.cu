#include "volrend/n3tree.hpp"
#include "volrend/cuda/common.cuh"

#include <limits>
#include <cstdio>
#include <cassert>

namespace volrend {
void N3Tree::load_cuda() {
    if (device.data != nullptr) cuda(Free(device.data));
    if (device.child != nullptr) cuda(Free(device.child));
    if (device.extra != nullptr) cuda(Free(device.extra));
    const size_t data_sz = (size_t) capacity * N3_ * data_dim * sizeof(half);
    const size_t child_sz = (size_t) capacity * N3_ * sizeof(int32_t);
    cuda(Malloc((void**)&device.data, data_sz));
    cuda(Malloc((void**)&device.child, child_sz));
    if (device.offset == nullptr) {
        cuda(Malloc((void**)&device.offset, 3 * sizeof(float)));
    }
    if (device.scale == nullptr) {
        cuda(Malloc((void**)&device.scale, 3 * sizeof(float)));
    }
    cuda(MemcpyAsync(device.child, child_.data<int32_t>(), child_sz,
                hipMemcpyHostToDevice));
    const half* data_ptr = this->data_.data<half>();
    cuda(MemcpyAsync(device.data, data_ptr, data_sz,
                hipMemcpyHostToDevice));
    cuda(MemcpyAsync(device.offset, offset.data(), 3 * sizeof(float),
                hipMemcpyHostToDevice));
    cuda(MemcpyAsync(device.scale, scale.data(), 3 * sizeof(float),
                hipMemcpyHostToDevice));
    if (extra_.data_holder.size()) {
        cuda(Malloc((void**)&device.extra, extra_.data_holder.size()));
        cuda(MemcpyAsync(device.extra, extra_.data<float>(),
                    extra_.data_holder.size(),
                    hipMemcpyHostToDevice));
    } else {
        device.extra = nullptr;
    }
    cuda_loaded_ = true;
}

void N3Tree::free_cuda() {
    if (device.data != nullptr) cuda(Free(device.data));
    if (device.child != nullptr) cuda(Free(device.child));
    if (device.offset != nullptr) cuda(Free(device.offset));
    if (device.scale != nullptr) cuda(Free(device.scale));
    if (device.extra != nullptr) cuda(Free(device.extra));
}
}  // namespace volrend

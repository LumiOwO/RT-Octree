#include "volrend/cuda/common.cuh"

#include <stdlib.h>
#include <stdio.h>

namespace volrend {

hipError_t cuda_assert(const hipError_t code, const char* const file,
                        const int line, const bool abort) {
    if (code != hipSuccess) {
        fprintf(stderr, "cuda_assert: %s %s %d\n", hipGetErrorString(code),
                file, line);

        if (abort) {
            hipDeviceReset();
            exit(code);
        }
    }

    return code;
}

}  // namespace volrend
